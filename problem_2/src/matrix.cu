#include "hip/hip_runtime.h"
#include "matrix.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <logger.h>
#include <stdlib.h>
#define CUDA_IDX2C(i,j,col_dim) (((i)*(col_dim))+(j))
#define CUBLAS_IDX2C(i,j,ld) (((j)*(ld))+(i))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

static const char *_cudaGetErrorEnum(hipblasStatus_t error);

__global__ void matrix_multiply(const float *lh_mat, const float *rh_mat, float *res_mat,
								const int lh_row, const int lh_col, 
								const int rh_row, const int rh_col)
{	
	int final_row = (blockIdx.x * blockDim.x) + threadIdx.x;
	int final_col = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (final_row >= lh_row || final_col >= rh_col)
	{
		return;
	}
	for(std::size_t k = 0; k < lh_col; ++k)
	{
		res_mat[CUDA_IDX2C(final_row, final_col, rh_col)] += lh_mat[CUDA_IDX2C(final_row, k, lh_col)] * rh_mat[CUDA_IDX2C(k, final_col, rh_col)];
	}
}

#define CHECK_ERR(x) if (x != hipSuccess) {						\
	err_logln("Cuda error caught! Error: ", hipGetErrorString(x)); \
	goto free;														\
}																	\

float *fl_cuda_matrix_multiply(float *lh_mat, float *rh_mat,
	const std::size_t lh_row, const std::size_t lh_col, const std::size_t rh_row, const std::size_t rh_col)
{
	const std::size_t n = rh_row * lh_col;
	const std::size_t size = n * sizeof(float);
	float *result_matrix = (float *)malloc(size);
	if (result_matrix == NULL)
	{
		err_logln("Error allocating host memory!%s", "");
		return NULL;
	}
	float *dev_rh_mat, *dev_lh_mat, *dev_res_mat;
	hipError_t error_stat;
	if ((error_stat = hipMalloc(&dev_rh_mat, rh_row*rh_col*sizeof(float))) != hipSuccess) {
		err_logln("Error allocating device memory! Error: %s", hipGetErrorString(error_stat));
		free(result_matrix);
		return NULL;
	}
	if ((error_stat = hipMalloc(&dev_lh_mat, lh_row*lh_col*sizeof(float))) != hipSuccess) {
		err_logln("Error allocating device memory! Error: %s", hipGetErrorString(error_stat));
		free(result_matrix);
		hipFree(dev_rh_mat);
		return NULL;
	}
	if ((error_stat = hipMalloc(&dev_res_mat, size)) != hipSuccess) {
		err_logln("Error allocating device memory! Error: %s", hipGetErrorString(error_stat));
		free(result_matrix);
		hipFree(dev_rh_mat);
		hipFree(dev_lh_mat);
		return NULL;
	}
	error_stat = hipMemcpy(dev_rh_mat, rh_mat, rh_row*rh_col * sizeof(float), hipMemcpyHostToDevice);
	CHECK_ERR(error_stat);
	error_stat = hipMemcpy(dev_lh_mat, lh_mat, lh_row*lh_col * sizeof(float), hipMemcpyHostToDevice);
	CHECK_ERR(error_stat);
	dim3 threads_per_block(32, 32);
	dim3 numb_blocks(lh_row/threads_per_block.x + 1, rh_col/threads_per_block.y + 1);
	matrix_multiply <<<numb_blocks, threads_per_block>>> (dev_lh_mat, dev_rh_mat, dev_res_mat, lh_row, lh_col, rh_row, rh_col);
	error_stat = hipMemcpy(result_matrix, dev_res_mat, size, hipMemcpyDeviceToHost);
	CHECK_ERR(error_stat);
cuda_free:
	if ((error_stat = hipFree(dev_rh_mat)) != hipSuccess) {
		err_logln("Error freeing device memory! Error: %s", hipGetErrorString(error_stat));
	}
	if ((error_stat = hipFree(dev_lh_mat)) != hipSuccess) {
		err_logln("Error freeing device memory! Error: %s", hipGetErrorString(error_stat));
	}
	if ((error_stat = hipFree(dev_res_mat)) != hipSuccess) {
		err_logln("Error freeing device memory! Error: %s", hipGetErrorString(error_stat));
	}
	return result_matrix;
free:
	free(result_matrix);
	goto cuda_free;
}


/*TODO: Complete*/
float *fl_cublas_matrix_multiply(float *lh_mat, float *rh_mat,
	std::size_t lh_row, std::size_t lh_col, std::size_t rh_row, std::size_t rh_col)
{
	hipError_t cuda_stat;
	hipblasStatus_t status;
	hipblasHandle_t handle;
	std::size_t n = lh_row * rh_col;
	std::size_t size = n * sizeof(float);
	float *ret_result = (float *)malloc(size);
	if (ret_result == NULL)
	{
		err_logln("Error allocating host memory.%s", "");
		return NULL;
	}
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		err_logln("Error initializing " GRN("cuBLAS") " context!");
		return NULL;
	}
	float *d_ret_result = NULL; 
	float *dev_ptr_lh = NULL;
	float *dev_ptr_rh = NULL;
	if ((cuda_stat = hipMalloc(&d_ret_result, size)) != hipSuccess) {
			err_logln("Cuda error caught! Error: ", hipGetErrorString(cuda_stat));
			goto destroy;														
	}
	if ((cuda_stat = hipMalloc(&dev_ptr_lh, sizeof(float)*lh_row*lh_col)) != hipSuccess) {
		err_logln("Cuda error caught! Error: ", hipGetErrorString(cuda_stat));
		hipFree(d_ret_result);
		goto destroy;
	}
	if ((cuda_stat = hipMalloc(&dev_ptr_rh, sizeof(float)*rh_row*rh_col)) != hipSuccess) {
		err_logln("Cuda error caught! Error: ", hipGetErrorString(cuda_stat));
		hipFree(d_ret_result);
		hipFree(dev_ptr_lh);
		goto destroy;
	}
	if ((cuda_stat = hipMemcpy(dev_ptr_lh, lh_mat, sizeof(float)*lh_row*lh_col, hipMemcpyHostToDevice)) != hipSuccess)
	{
		err_logln("Cuda error caught! Error: ", hipGetErrorString(cuda_stat));
		goto free;
	}
	if ((cuda_stat = hipMemcpy(dev_ptr_rh, rh_mat, sizeof(float)*rh_row*rh_col, hipMemcpyHostToDevice)) != hipSuccess)
	{
		err_logln("Cuda error caught! Error: ", hipGetErrorString(cuda_stat));
		goto free;
	}
	float alpha = 1.0f;
	float beta = 0.0f;
	status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rh_col, lh_row, lh_col, &alpha, dev_ptr_rh, rh_col, dev_ptr_lh, lh_col, &beta, d_ret_result, MIN(rh_col, lh_col));
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		err_logln("Error multiplying matrices! Error code: %s", _cudaGetErrorEnum(status));
		goto free;
	}
	if ((cuda_stat = hipMemcpy(ret_result, d_ret_result, size, hipMemcpyDeviceToHost)) != hipSuccess)
	{
		err_logln("Cuda error caught! Error: ", hipGetErrorString(cuda_stat));
		goto free;
	}
cuda_free:
	hipFree(dev_ptr_lh);
	hipFree(dev_ptr_rh);
	hipFree(d_ret_result);
destroy:
	hipblasDestroy(handle);
	return ret_result;
free:
	free(ret_result);
	goto cuda_free;
}

/* cuBLAS API errors */
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}