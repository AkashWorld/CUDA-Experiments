#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <logger.h>
#include <stdlib.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static const char *_cudaGetErrorEnum(hipblasStatus_t error);

__global__ void matrix_multiply(float *rh_mat, float *lh_mat, float *res_mat)
{

}

#define CHECK_ERR(x) if (x != hipSuccess) {						\
	err_logln("Cuda error caught! Error: ", hipGetErrorString(x)); \
	goto free;														\
}																	\

/*TODO: Complete*/
float *fl_cuda_matrix_multiply(float *rh_mat, float *lh_mat,
	std::size_t lh_row, std::size_t lh_col, std::size_t rh_row, std::size_t rh_col)
{
	const std::size_t n = rh_row * rh_col;
	const std::size_t size = n * sizeof(float);
	float *result_matrix = (float *)malloc(size);
	if (result_matrix == NULL)
	{
		err_logln("Error allocating host memory!%s", "");
		return NULL;
	}
	float *dev_rh_mat, *dev_lh_mat, *dev_res_mat;
	hipError_t error_stat;
	if ((error_stat = hipMalloc(&dev_rh_mat, size)) != hipSuccess) {
		err_logln("Error allocating device memory! Error: %s", hipGetErrorString(error_stat));
		free(result_matrix);
		return NULL;
	}
	if ((error_stat = hipMalloc(&dev_lh_mat, size)) != hipSuccess) {
		err_logln("Error allocating device memory! Error: %s", hipGetErrorString(error_stat));
		free(result_matrix);
		hipFree(dev_rh_mat);
		return NULL;
	}
	if ((error_stat = hipMalloc(&dev_res_mat, size)) != hipSuccess) {
		err_logln("Error allocating device memory! Error: %s", hipGetErrorString(error_stat));
		free(result_matrix);
		hipFree(dev_rh_mat);
		hipFree(dev_lh_mat);
		return NULL;
	}
	error_stat = hipMemcpy(dev_rh_mat, rh_mat, size, hipMemcpyHostToDevice);
	CHECK_ERR(error_stat);
	error_stat = hipMemcpy(dev_lh_mat, lh_mat, size, hipMemcpyHostToDevice);
	CHECK_ERR(error_stat);
	
	

	error_stat = hipMemcpy(result_matrix, dev_res_mat, size, hipMemcpyDeviceToHost);
	CHECK_ERR(error_stat);
free:
	if ((error_stat = hipFree(dev_rh_mat)) != hipSuccess) {
		err_logln("Error freeing device memory! Error: %s", hipGetErrorString(error_stat));
	}
	if ((error_stat = hipFree(dev_lh_mat)) != hipSuccess) {
		err_logln("Error freeing device memory! Error: %s", hipGetErrorString(error_stat));
	}
	if ((error_stat = hipFree(dev_res_mat)) != hipSuccess) {
		err_logln("Error freeing device memory! Error: %s", hipGetErrorString(error_stat));
	}
	return result_matrix;
}

/*TODO: Complete*/
float *fl_cublas_matrix_multiply(float *rh_mat, float *lh_mat, 
		std::size_t lh_row, std::size_t lh_col, std::size_t rh_row, std::size_t rh_col)
{
	std::size_t n = lh_row * rh_col;
	std::size_t size = n * sizeof(float);
	float *ret_result = (float *)malloc(size);
	if (ret_result == NULL)
	{
		err_logln("Error allocating host memory.%s", "");
		return NULL;
	}
	hipError_t cuda_stat;
	hipblasStatus_t status;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		err_logln("Error initializing " GRN("cuBLAS") " context!");
		return NULL;
	}
	float *dev_ptr_lh, *dev_ptr_rh;
	cuda_stat = hipMalloc(&dev_ptr_lh, sizeof(float)*n);
	if (cuda_stat != hipSuccess)
	{
		err_logln("Error allocating " GRN("device") "memory!, Error code: %d", cuda_stat);
		return NULL;
	}
	cuda_stat = hipMalloc(&dev_ptr_rh, sizeof(float)*n);
	if (cuda_stat != hipSuccess)
	{
		err_logln("Error allocating " GRN("device") "memory! Error code: %d", cuda_stat);
		return NULL;
	}
	status = hipblasSetVector(n, sizeof(float), rh_mat, sizeof(float), dev_ptr_rh, sizeof(float));
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		err_logln("Error copying matrix into device! Error code: %s", _cudaGetErrorEnum(status));
		return NULL;
	}
	status = hipblasSetVector(n, sizeof(float), lh_mat, sizeof(float), dev_ptr_lh, sizeof(float));
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		err_logln("Error copying matrix into device! Error code: %s", _cudaGetErrorEnum(status));
		return NULL;
	}
	status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, lh_row, lh_col, rh_col, NULL, dev_ptr_lh, lh_row,
			dev_ptr_rh, rh_row, NULL, ret_result, lh_row);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		err_logln("Error multiplying matrices! Error code: %s", _cudaGetErrorEnum(status));
		return NULL;
	}
	hipblasDestroy(handle);
	return ret_result;
}

// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}