#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <logger.h>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static const char *_cudaGetErrorEnum(hipblasStatus_t error);

float *fl_cublas_matrix_multiply(float *rh_mat, float *lh_mat, 
		std::size_t lh_row, std::size_t lh_col, std::size_t rh_row, std::size_t rh_col)
{
	float *ret_result;
	std::size_t n = lh_row * rh_col;
	hipError_t cuda_stat;
	hipblasStatus_t status;
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		err_logln("Error initializing " GRN("cuBLAS") " context!");
		return NULL;
	}
	float *dev_ptr_lh, *dev_ptr_rh;
	cuda_stat = hipMalloc(&dev_ptr_lh, sizeof(float)*n);
	if (cuda_stat != hipSuccess)
	{
		err_logln("Error allocating " GRN("device") "memory!, Error code: %d", cuda_stat);
		return NULL;
	}
	cuda_stat = hipMalloc(&dev_ptr_rh, sizeof(float)*n);
	if (cuda_stat != hipSuccess)
	{
		err_logln("Error allocating " GRN("device") "memory! Error code: %d", cuda_stat);
		return NULL;
	}
	status = hipblasSetVector(n, sizeof(float), rh_mat, sizeof(float), dev_ptr_rh, sizeof(float));
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		err_logln("Error copying matrix into device! Error code: %s", _cudaGetErrorEnum(status));
		return NULL;
	}
	status = hipblasSetVector(n, sizeof(float), lh_mat, sizeof(float), dev_ptr_lh, sizeof(float));
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		err_logln("Error copying matrix into device! Error code: %s", _cudaGetErrorEnum(status));
		return NULL;
	}
	status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, lh_row, lh_col, rh_col, NULL, dev_ptr_lh, lh_row,
			dev_ptr_rh, rh_row, NULL, ret_result, lh_row);
	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		err_logln("Error multiplying matrices! Error code: %s", _cudaGetErrorEnum(status));
		return NULL;
	}
	hipblasDestroy(handle);
	return ret_result;
}

// cuBLAS API errors
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
	switch (error)
	{
	case HIPBLAS_STATUS_SUCCESS:
		return "HIPBLAS_STATUS_SUCCESS";

	case HIPBLAS_STATUS_NOT_INITIALIZED:
		return "HIPBLAS_STATUS_NOT_INITIALIZED";

	case HIPBLAS_STATUS_ALLOC_FAILED:
		return "HIPBLAS_STATUS_ALLOC_FAILED";

	case HIPBLAS_STATUS_INVALID_VALUE:
		return "HIPBLAS_STATUS_INVALID_VALUE";

	case HIPBLAS_STATUS_ARCH_MISMATCH:
		return "HIPBLAS_STATUS_ARCH_MISMATCH";

	case HIPBLAS_STATUS_MAPPING_ERROR:
		return "HIPBLAS_STATUS_MAPPING_ERROR";

	case HIPBLAS_STATUS_EXECUTION_FAILED:
		return "HIPBLAS_STATUS_EXECUTION_FAILED";

	case HIPBLAS_STATUS_INTERNAL_ERROR:
		return "HIPBLAS_STATUS_INTERNAL_ERROR";
	}

	return "<unknown>";
}