#include "hip/hip_runtime.h"
/* created by Suva Shahria*/


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <queue>
#include <time.h>
#include <cstring>

#include <cstdlib>
#include <iostream>
#include <time.h>


using namespace std;


/*
Given a vertex, different thread checks each vertex to see if there is an edge
connecting the vertices.
*/

__global__
void cuda_bfs(int v, int idx, int * dmat, bool * d_visited, int * d_push) {



	int index = idx * v;

	if (threadIdx.x < v) {

		if (dmat[index + threadIdx.x] && !d_visited[threadIdx.x]) {
			//printf("%d-----%d\n", index + threadIdx.x, threadIdx.x);
			//	printf("%d\n", threadIdx.x);
			d_visited[threadIdx.x] = true;
			d_push[threadIdx.x] = 1;
		}
	}


	/*if (threadIdx.x < v) {
	if (visited[threadIdx.x] == true) {
	printf("%d\n", threadIdx.x);

	}

	}
	/*if (threadIdx.x < v*v) {
	if (dmat[threadIdx.x] == 1) {

	printf("%d\n", threadIdx.x);
	}
	}
	*/
}

int main(int arg, char** argv) {
	int* mat;
	int i;
	int v;

	char single;

	if (arg != 3)
	{
		printf("usage: ./out size, starting_index example: ./t 5 2 \n");

		return -1;
	}
	v = atoi(argv[1]);
	int start = atoi(argv[2]);

	if (start >= v || start<0) {
		printf("start index is out of bounds \n");
		return -1;

	}
	FILE *pToFile = fopen("graph.txt", "r");


	i = 0;



	//


	mat = (int*)malloc(v *v * sizeof(int));
	

	//read from mygraph.txt
	while ((single = fgetc(pToFile)) != EOF) {


		if (single != '\n') {

			if (single == '1') {
				// cout << i << endl;
				mat[i] = 1;
				//cout << mat[i] << endl;
			}
			else {
				mat[i] = 0;
			}
			i++;
		}

	}


	fclose(pToFile);





	int * dmat;
	// create device objects

	hipMalloc((void**)&dmat, sizeof(int) * v*v);
	hipMemcpy((void*)dmat, (void*)mat, sizeof(int)*v*v, hipMemcpyHostToDevice);



	bool* visited = (bool*)malloc(v * sizeof(bool));   //visited
	bool* d_visited;

	for (int i = 0; i < v; i++) {
		visited[i] = false;
	}
	visited[start] = true;


	hipMalloc((void**)&d_visited, sizeof(bool) * v);
	hipMemcpy((void*)d_visited, (void*)visited, sizeof(bool)*v, hipMemcpyHostToDevice);




	queue<int> q;						//queue
	q.push(start);

	int* h_push = (int*)malloc(v * sizeof(int));			//h push
	int* d_push;
	for (i = 0; i < v; i++) {
		h_push[i] = 0;
	}



	hipMalloc((void**)&d_push, sizeof(int) * v);
	hipMemcpy((void*)d_push, (void*)h_push, sizeof(int)*v, hipMemcpyHostToDevice);

	hipEvent_t st1, stop;
	hipEventCreate(&st1);
	hipEventCreate(&stop);
	float milliseconds = 0;

	int j = 0;;
	hipEventRecord(st1);

	// once vertex is found to be a neighbor adds it to end
	while (!q.empty()) {
		for (i = 0; i < v; i++) {
			h_push[i] = 0;
		}


		hipMemcpy((void*)d_push, (void*)h_push, sizeof(int)*v, hipMemcpyHostToDevice);
		//	hipMemcpy((void*)d_visited, (void*)visited, sizeof(bool)*v, hipMemcpyHostToDevice);
		i = q.front();
		cout << q.front() << " ";
		q.pop();

		cuda_bfs << <1, v >> >(v, i, dmat, d_visited, d_push);

		hipMemcpy((void*)h_push, (void*)d_push, sizeof(int) * v, hipMemcpyDeviceToHost);
		//	hipMemcpy((void*)d_visited, (void*)visited, sizeof(bool) * v, hipMemcpyDeviceToHost);
		//	cout << h_push[1];
		for (j = 0; j < v; j++) {
			if (h_push[j] == 1) {
				//cout << j;

				q.push(j);
			}

		}

	}
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, st1, stop);

	cout << endl << milliseconds << " ms" <<endl;
	hipFree(dmat);
	hipFree(d_visited);
	hipFree(d_push);
	//

	//cuda_bfs << <1, v >> >(v,i, dmat,d_visited, d_push);
	return 0;
}
